#include "hip/hip_runtime.h"
#include "dependencies.cuh"
#include "particle.cuh"
#include "structs.cuh"
#include "kernel_helper.cuh"

//Solvers

__device__ void fftsolver_device(hipfftDoubleComplex* freq, Config cfg){
	size_t
		i = blockIdx.x * blockDim.x + threadIdx.x,
		j = blockIdx.y * blockDim.y + threadIdx.y,
		k = blockIdx.z * blockDim.z + threadIdx.z;
	if (i >= cfg.n.x || j >= cfg.n.y || k >= cfg.n.w) { return; }
	size_t offset = i + cfg.n.x * (j + cfg.n.y * k);

	if (i > cfg.n.x / 2)
		i = cfg.n.x - i;
	if (j > cfg.n.y / 2)
		j = cfg.n.y - j;
	if (k > cfg.n.z / 2)
		k = cfg.n.z - k;

	double K_sqrd = cfg.solve.kxt * i*i + cfg.solve.kyt * j*j + cfg.solve.kzt * k*k;
	if (i + j + k == 0)
		K_sqrd = 1.0;

	double scale = cfg.solve.constant_factor / K_sqrd;
	hipfftDoubleComplex val = freq[offset];
	val.x *= scale;
	val.y *= scale;
	freq[offset] = val;
}
//FFT-solver
__global__ void fftsolver_kernel(hipfftDoubleComplex* freq, Config cfg) {
	fftsolver_device(freq, cfg);
}
//SOR-solver
__global__ void sorsolver_kernel(double* phi, Config cfg, size_t flag) {
	size_t
		i = blockIdx.x * blockDim.x + threadIdx.x,
		j = blockIdx.y * blockDim.y + threadIdx.y,
		k = 2 * (blockIdx.z * blockDim.z + threadIdx.z) + (i + j + flag) % 2;
	if (i >= cfg.n.x || j >= cfg.n.y || k >= cfg.n.z) { return; }
	
	size_t
		l = (i != 0 ? i - 1 : i),
		r = (i != cfg.n.x - 1 ? i + 1 : i),

		d = (j != 0 ? j - 1 : j) * cfg.n.x,
		u = (j != cfg.n.y - 1 ? j + 1 : j) * cfg.n.x,

		f = (k != 0 ? k - 1 : k) * cfg.n.x * cfg.n.y,
		b = (k != cfg.n.z - 1 ? k + 1 : k) * cfg.n.x * cfg.n.y;

	j = j * cfg.n.x;
	k = k * cfg.n.x * cfg.n.y;
	
	double
		center	= phi[i + j + k],
		left	= phi[l + j + k],
		right	= phi[r + j + k],
		down	= phi[i + d + k],
		up		= phi[i + u + k],
		front	= phi[i + j + f],
		back	= phi[i + j + b],
		tmp, val;

	tmp = (left + right + down + up + front + back) / 6;
	val = center + cfg.omega * (tmp - center);

	phi[i + j + k] = val;
}
//Saturate phi with initial values from rho
__global__ void sorinit_kernel(double* phi, double* rho, Config cfg) {
	size_t
		i = blockIdx.x * blockDim.x + threadIdx.x,
		j = blockIdx.y * blockDim.y + threadIdx.y,
		k = blockIdx.z * blockDim.z + threadIdx.z;
	if (i >= cfg.n.x || j >= cfg.n.y || k >= cfg.n.z) { return; }
	size_t offset = i + cfg.n.x * (j + cfg.n.y * k);

	double
		h2 = cfg.l.x / cfg.n.x + cfg.l.y / cfg.n.y + cfg.l.z / cfg.n.z;

	phi[offset] = rho[offset] * h2 / (cfg.epsilon *  6);
}

//Electric field calculation
__global__ void electricfield_kernel(double* phi, double4* E, Config cfg) {
	size_t
		i = blockIdx.x * blockDim.x + threadIdx.x,
		j = blockIdx.y * blockDim.y + threadIdx.y,
		k = blockIdx.z * blockDim.z + threadIdx.z;
	if (i >= cfg.n.x || j >= cfg.n.y || k >= cfg.n.z) { return; }

	size_t
		left	= (i != 0 ? -1 : 0),
		down	= (j != 0 ? -1 : 0) * cfg.n.x,
		front	= (k != 0 ? -1 : 0) * cfg.n.x * cfg.n.y,
		right	= (i != cfg.n.x - 1 ? 1 : 0),
		up		= (j != cfg.n.y - 1 ? 1 : 0) * cfg.n.x,
		back	= (k != cfg.n.z - 1 ? 1 : 0) * cfg.n.x * cfg.n.y;

	j *= cfg.n.x;
	k *= cfg.n.x * cfg.n.y;
	size_t index = i + j + k;

	double
		x = phi[index + left]	- phi[index + right],
		y = phi[index + up]		- phi[index + down],
		z = phi[index + front]	- phi[index + back];
	E[index] = make_double4(x, y, z, 0.0);
}

//Particle updates and charge distribution
__global__ void particleUpdate_kernel(Particle* particles, double4* E, double* rho, Config cfg) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= cfg.particles) { return; }

	double3 h = div_v(cfg.l, cfg.n);
	Particle p = particles[idx];
	ParticleCfg pcfg;
	pCfg(p, cfg, h, &pcfg);

	double3 a;
	a.x = E[pcfg.o.fll].x * pcfg.w.zyx
		+ E[pcfg.o.flr].x * pcfg.w.zya
		+ E[pcfg.o.ful].x * pcfg.w.zbx
		+ E[pcfg.o.fur].x * pcfg.w.zba
		+ E[pcfg.o.bll].x * pcfg.w.cyx
		+ E[pcfg.o.blr].x * pcfg.w.cya
		+ E[pcfg.o.bul].x * pcfg.w.cbx
		+ E[pcfg.o.bur].x * pcfg.w.cba,

	a.y = E[pcfg.o.fll].y * pcfg.w.zyx
		+ E[pcfg.o.flr].y * pcfg.w.zya
		+ E[pcfg.o.ful].y * pcfg.w.zbx
		+ E[pcfg.o.fur].y * pcfg.w.zba
		+ E[pcfg.o.bll].y * pcfg.w.cyx
		+ E[pcfg.o.blr].y * pcfg.w.cya
		+ E[pcfg.o.bul].y * pcfg.w.cbx
		+ E[pcfg.o.bur].y * pcfg.w.cba,

	a.z	= E[pcfg.o.fll].z * pcfg.w.zyx
		+ E[pcfg.o.flr].z * pcfg.w.zya
		+ E[pcfg.o.ful].z * pcfg.w.zbx
		+ E[pcfg.o.fur].z * pcfg.w.zba
		+ E[pcfg.o.bll].z * pcfg.w.cyx
		+ E[pcfg.o.blr].z * pcfg.w.cya
		+ E[pcfg.o.bul].z * pcfg.w.cbx
		+ E[pcfg.o.bur].z * pcfg.w.cba;

	double c_by_h = cfg.charge_by_mass / vol(h);

	a = mul_s(a, c_by_h);
	p.velocity = add_v(p.velocity, mul_s(a, cfg.ts));

	p.position = add_v(p.position, mul_s(p.velocity, cfg.ts));

	particles[idx] = p;

	pCfg(p, cfg, h, &pcfg);

	//Add the particle's contribution to each neighbouring vertices.
	atomicAdd(&rho[pcfg.o.fll], pcfg.w.zyx * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.flr], pcfg.w.zya * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.ful], pcfg.w.zbx * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.fur], pcfg.w.zba * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.bll], pcfg.w.cyx * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.blr], pcfg.w.cya * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.bul], pcfg.w.cbx * cfg.rho_k);
	atomicAdd(&rho[pcfg.o.bur], pcfg.w.cba * cfg.rho_k);
}