#include "hip/hip_runtime.h"
#include "dependencies.cuh"
#include "structs.cuh"
#include "simplified_kernels.cuh"
#include "pic_utils.cuh"
#include "cfg.cuh"
#include "Tracer.cuh"
#include "ArrayPrinter.cuh"
#include "Particle.cuh"

//#define TRACE

namespace pic{
	double* d_rhophi;
	double4* d_efreq;
	Particle* d_particles;

	hipfftResult_t setupCufft(hipStream_t streamCalc, hipfftHandle* plan, hipfftHandle* iplan, Config cfg) {
		cufftChk(hipfftCreate(plan));
		hipfftResult_t res = hipfftPlan3d(plan, cfg.n.z, cfg.n.y, cfg.n.x, HIPFFT_D2Z);
		hipfftSetStream(*plan, streamCalc);

		cufftChk(hipfftCreate(iplan));
		hipfftResult_t ires = hipfftPlan3d(iplan, cfg.n.w, cfg.n.y, cfg.n.x, HIPFFT_Z2D);
		hipfftSetStream(*iplan, streamCalc);

		//#define PIC_CUFFT_CALLBACK_SETUP
#ifdef PIC_CUFFT_CALLBACK_SETUP
		__device__ void (*device_solver_callback_ptr)(hipfftDoubleComplex* freq, Config cfg) = fftsolve;
		void (*host_solver_callback_ptr)(hipfftDoubleComplex* freq, Config cfg);
		hipMemcpyFromSymbol(&host_solver_callback_ptr, HIP_SYMBOL(device_solver_callback_ptr), sizeof(host_solver_callback_ptr));
		hipfftXtSetCallback(host_solver_callback_ptr);
#endif

		if (res != HIPFFT_SUCCESS) {
			cufftChk(res);
			cufftChk(ires);
			cufftChk(hipfftDestroy(*plan));
			cufftChk(hipfftDestroy(*iplan));
			return res;
		}
		else if (ires != HIPFFT_SUCCESS){
			cufftChk(res);
			cufftChk(ires);
			cufftChk(hipfftDestroy(*plan));
			cufftChk(hipfftDestroy(*iplan));
			return ires;
		}

		return HIPFFT_SUCCESS;
	}

	void fftsolver_host(Config cfg, hipfftHandle plan, hipfftHandle iplan){
		cufftChk(hipfftExecD2Z(plan, (hipfftDoubleReal *)d_rhophi, (hipfftDoubleComplex *)d_efreq));
		fftsolver_kernel <<<cfg.exec_cfg.nbfreq, cfg.exec_cfg.tbg >>>((hipfftDoubleComplex *)d_efreq, cfg);
		errCheck();
		cudaChk(hipMemset(d_rhophi, 0, cfg.n.x * cfg.n.y * cfg.n.z * sizeof(double)));
		cufftChk(hipfftExecZ2D(iplan, (hipfftDoubleComplex *)d_efreq, (hipfftDoubleReal *)d_rhophi));
	}

	void sorsolver_host(hipStream_t streamCalc, Config cfg){
		sorinit_kernel << <cfg.exec_cfg.nbg, cfg.exec_cfg.tbg, 0, streamCalc >> >(d_rhophi, d_rhophi, cfg);
		errCheck();
		size_t shared_size = 2 * cfg.exec_cfg.tbg.x * 2 * cfg.exec_cfg.tbg.y * 2 * cfg.exec_cfg.tbg.z * sizeof(double);

		for (int it_sor = 0; it_sor < cfg.sor_iterations; it_sor++) {
			sorsolver_shared << <cfg.exec_cfg.nbg, cfg.exec_cfg.tbg, shared_size, streamCalc >> >(d_rhophi, cfg, 0);
			errCheck();
			//sorsolver_kernel << <cfg.exec_cfg.nbsor, cfg.exec_cfg.tbg, shared_size, streamCalc >> >(d_rhophi, cfg, 1);
			//errCheck();
		}
	}

	void initParticles(Particle* d_particles, Config cfg) {
		Particle* h_particles = new Particle[cfg.particles];


		for (int i = 0; i < cfg.particles; i++) {
			h_particles[i] = Particle(cfg);
		}
		hipMemcpy(d_particles, h_particles, cfg.particles * sizeof(Particle), hipMemcpyHostToDevice);
		delete h_particles;
	}

	int run(Config cfg, bool fft = false) {
		//setup
		hipfftHandle plan, iplan;

		//Create streams
		hipStream_t streamCalc, streamTrace;
		cudaChk(hipStreamCreate(&streamCalc));
		cudaChk(hipStreamCreate(&streamTrace));

		if (setupCufft(streamCalc, &plan, &iplan, cfg) != HIPFFT_SUCCESS)
			return EXIT_FAILURE;
		
		//Allocate arrays
		cudaChk(hipMalloc(&d_particles, cfg.particles * sizeof(Particle)));
		initParticles(d_particles, cfg);
		Tracer<Particle> *tf = new Tracer<Particle>(streamTrace, (cfg.iterations + 1), cfg.particles, d_particles, "_particles");
		tf->appendTrace();

		size_t
			rhophiSize = cfg.n.x * cfg.n.y * cfg.n.z * sizeof(double),
			efreqSize = cfg.n.x * cfg.n.y * cfg.n.z * sizeof(double4);
		cudaChk(hipMalloc(&d_rhophi, rhophiSize));
		cudaChk(hipMalloc(&d_efreq, efreqSize));
		ArrayPrinter<double> *rhophiPrinter = new ArrayPrinter<double>(cfg.iterations, cfg.n.x, cfg.n.y, cfg.n.z, d_rhophi, "rhopi");
		ArrayPrinter<double4> *efreqPrinter = new ArrayPrinter<double4>(cfg.iterations, cfg.n.x, cfg.n.y, cfg.n.z, d_efreq, "efreq");

		//loop
		cudaChk(hipMemset(d_efreq, 0, efreqSize));
		int iteration = 0;
		while (iteration < cfg.iterations) {

			hipStreamSynchronize(streamTrace);
			cudaChk(hipMemset(d_rhophi, 0, rhophiSize));
			particleUpdate_kernel << <cfg.exec_cfg.nbp, cfg.exec_cfg.tbp, 0, streamCalc >> >(d_particles, d_efreq, d_rhophi, cfg); errCheck();
			
			hipStreamSynchronize(streamCalc);
			tf->appendTrace(); errCheck();	

			rhophiPrinter->appendValues(); errCheck();
						
			cudaChk(hipMemset(d_efreq, 0, efreqSize));
			if (fft)
				fftsolver_host(cfg, plan, iplan);
			else
				sorsolver_host(streamCalc, cfg);

			cudaChk(hipMemset(d_efreq, 0, efreqSize));
			electricfield_kernel << <cfg.exec_cfg.nbg, cfg.exec_cfg.tbg, 0, streamCalc >> >(d_rhophi, d_efreq, cfg); errCheck();

			efreqPrinter->appendValues(); errCheck();
	
			iteration++;
		}
		tf->print();
		rhophiPrinter->print();
		efreqPrinter->print();
		delete tf;
		delete rhophiPrinter, efreqPrinter;

		//clean
		cudaChk(hipFree(d_rhophi));
		cudaChk(hipFree(d_efreq));
		cudaChk(hipFree(d_particles));
		cufftChk(hipfftDestroy(plan));
		cufftChk(hipfftDestroy(iplan));

		return EXIT_SUCCESS;
	}
}

int main(int argc, const char* argv[]){
	Config cfg = getConfig();
#ifndef TRACE
	cfg.iterations = 4;
#endif
	int res = pic::run(cfg, false);
	return res;
}
